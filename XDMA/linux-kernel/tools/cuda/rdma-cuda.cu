#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#define GL_GLEXT_PROTOTYPES
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include "../../xdma/cdev_rdma.h"

#include <GL/gl.h>
#include <GL/glext.h>
#include <GL/glut.h>

#define SURFACE_W	3840
#define SURFACE_H	2160
#define SURFACE_SIZE	(SURFACE_W * SURFACE_H)

#define INPUT_WIDTH_BYTES ((SURFACE_W * 20)/8)
#define INPUT_STRIDE_BYTES 0x4000
#define INPUT_BYTES_PER_SAMPLE 5

#define OFFSET(x, y)	(((y) * SURFACE_W) + x)
#define DATA(x, y)	(((y & 0xffff) << 16) | ((x) & 0xffff))

inline __device__ __host__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}

// round up n/m
inline int iDivUp(int n, int m)
{
    return (n + m - 1) / m;
}

extern "C" __global__ void fill_surface(uint32_t *output, uint32_t xor_val)
{
	unsigned int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;

	output[OFFSET(pos_x, pos_y)] = DATA(pos_x, pos_y) ^ xor_val;
}

extern "C" __global__ void convertYuv10ToRGBA(unsigned char *input, unsigned char *output, unsigned int width, unsigned int stride)
{
	unsigned int orig_pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int pos_x = ((blockIdx.x * blockDim.x) + threadIdx.x) * INPUT_BYTES_PER_SAMPLE;
	unsigned int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	// YUV 10bit is 20bit per pixel. We take 2 samples, so 40bit = 5 bytes worth of data, and output 2 RGBA samples, so 64 bit = 8 bytes of data
	// Pixels are shifted in batches of 512bit = 64 bytes, so the first bytes will the MSBs of that batch
	const unsigned int BatchSize = 64;
	unsigned int batchIndex = pos_x % BatchSize;
	unsigned int nextBatchOffset = ((pos_x / BatchSize) + 1) * BatchSize;
	int xOffset = (nextBatchOffset-INPUT_BYTES_PER_SAMPLE) - batchIndex;
	int offset = xOffset + pos_y * stride;
	const unsigned char inputPx0 = input[offset+0];
	const unsigned char inputPx1 = input[offset+1];
	const unsigned char inputPx2 = input[offset+2];
	const unsigned char inputPx3 = input[offset+3];
	const unsigned char inputPx4 = input[offset+4];
	
	float y0 = inputPx0 + (inputPx1 & 0x03) * 256.0f;
	float u = (inputPx1 & 0xFC) / 4.0f + (inputPx2 & 0x0F) * 64.0f;
	float y1 = (inputPx2 & 0xF0) / 16.0f + (inputPx3 & 0x3F) * 16.0f;
	float v = (inputPx3 & 0xC0) / 64.0f +  inputPx4 * 4.0f;
	
	const float4 px0 = make_float4( y0 + 1.140f * v,
					 y0 - 0.395f * u - 0.581f * v,
					 y0 + 2.032f * u, 255.0f );

	const float4 px1 = make_float4( y1 + 1.140f * v,
					 y1 - 0.395f * u - 0.581f * v,
					 y1 + 2.032f * u, 255.0f );
	
	const uchar4 rgb1 = make_uchar4(
				    	clamp(px0.x/4.0f, 0.0f, 255.0f),
				    	clamp(px0.y/4.0f, 0.0f, 255.0f),
		    			clamp(px0.z/4.0f, 0.0f, 255.0f),
					255.0f );
	const uchar4 rgb2 = make_uchar4( 
				    	clamp(px1.x/4.0f, 0.0f, 255.0f),
				    	clamp(px1.y/4.0f, 0.0f, 255.0f),
		    			clamp(px1.z/4.0f, 0.0f, 255.0f),
					255.0f );
						
	unsigned int outputStride = SURFACE_W * 4;	
	unsigned int outputXOffset = (orig_pos_x * 8);
	unsigned int outputOffset = outputXOffset + pos_y * outputStride;				
	output[outputOffset] = rgb1.x;
	output[outputOffset+1] = rgb1.y;
	output[outputOffset+2] = rgb1.z;
	output[outputOffset+3] = rgb1.w;
	output[outputOffset+4] = rgb2.x;
	output[outputOffset+5] = rgb2.y;
	output[outputOffset+6] = rgb2.z;
	output[outputOffset+7] = rgb2.w;
}

GLuint pbo = 0;
GLuint tex = 0;
struct hipGraphicsResource *cuda_pbo_resource;
uint32_t *src_d, *dst_d;

void drawTexture()
{
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, SURFACE_W, SURFACE_H, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
	glEnable(GL_TEXTURE_2D);
	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f); glVertex2f(0, 0);
	glTexCoord2f(0.0f, 1.0f); glVertex2f(0, SURFACE_H);
	glTexCoord2f(1.0f, 1.0f); glVertex2f(SURFACE_W, SURFACE_H);
	glTexCoord2f(1.0f, 0.0f); glVertex2f(SURFACE_W, 0);
	glEnd();
	glDisable(GL_TEXTURE_2D);
}

void render()
{
	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
	uchar4 *d_out = 0;
	size_t size = 0;
	hipGraphicsResourceGetMappedPointer((void **)&d_out, &size, cuda_pbo_resource);
	dim3 dimBlock(16, 16);
	dim3 dimGrid(iDivUp(INPUT_WIDTH_BYTES, INPUT_BYTES_PER_SAMPLE * dimBlock.x), iDivUp(SURFACE_H, dimBlock.y));
	convertYuv10ToRGBA<<<dimGrid, dimBlock>>>(reinterpret_cast<unsigned char*>(src_d), reinterpret_cast<unsigned char*>(d_out), INPUT_WIDTH_BYTES, INPUT_STRIDE_BYTES);
	hipDeviceSynchronize();
	//hipMemcpy(d_out, src_d, SURFACE_SIZE * 4, hipMemcpyDeviceToDevice);
	
	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}

void display()
{
	render();
	drawTexture();
	glutSwapBuffers();
	glutPostRedisplay();
}

void onKeyboardPress(unsigned char key, int x, int y)
{
	if (key == 'p')
	{
	    printf("onKeyboardPress %d\n", key);
	}
}

void initOpenGL(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(2160, 2160);
	glutInitWindowPosition(100, 100);
	glutCreateWindow("Title");
	glutDisplayFunc(display);
	glutKeyboardFunc(onKeyboardPress);
	gluOrtho2D(0, SURFACE_W, SURFACE_H, 0);
}

void initPixelBuffer()
{
	// PBO are a "source" of a texture, its buffer data can be "unpacked" to texture.
	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, SURFACE_SIZE * 4, NULL, GL_STREAM_DRAW);
	
	glGenTextures(1, &tex);
	glBindTexture(GL_TEXTURE_2D, tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_BORDER);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_BORDER);
	
	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, hipGraphicsRegisterFlagsWriteDiscard);
}

int main(int argc, char **argv)
{
	hipError_t ce;
	hipError_t cr;
	uint32_t *dst_cpu;
	uint32_t y, x;
	int fd, ret;
	unsigned int flag = 1;
	struct rdma_pin_cuda pin_params_src, pin_params_dst;
	struct rdma_h2c2h_dma dma_params;
	struct rdma_unpin_cuda unpin_params_src, unpin_params_dst;

	if (argc != 1) {
		fprintf(stderr, "usage: rdma-cuda\n");
		return 1;
	}
	
	initOpenGL(argc, argv);
	initPixelBuffer();
	
	fd = open("/dev/xdma1_rdma", O_RDWR);
	if (fd < 0) {
		perror("open() failed");
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipMalloc(&src_d, SURFACE_SIZE * sizeof(*src_d));
#else
	ce = hipHostAlloc(&src_d, SURFACE_SIZE * sizeof(*src_d),
		hipHostMallocDefault);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Allocation of src_d failed: %d\n", ce);
		return 1;
	}

	cr = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
		(hipDeviceptr_t)src_d);
	if (cr != hipSuccess) {
		fprintf(stderr, "hipPointerSetAttribute(src_d) failed: %d\n", cr);
		return 1;
	}

	pin_params_src.va = (__u64)src_d;
	pin_params_src.size = SURFACE_SIZE * sizeof(*src_d);
	ret = ioctl(fd, RDMA_IOC_PIN, &pin_params_src);
	if (ret != 0) {
		fprintf(stderr, "ioctl(RDMA_IOC_PIN src) failed: ret=%d errno=%d\n", ret, errno);
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipMalloc(&dst_d, SURFACE_SIZE * sizeof(*dst_d));
#else
	ce = hipHostAlloc(&dst_d, SURFACE_SIZE * sizeof(*dst_d),
		hipHostMallocDefault);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Allocation of dst_d failed: %d\n", ce);
		return 1;
	}

	cr = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
		(hipDeviceptr_t)dst_d);
	if (cr != hipSuccess) {
		fprintf(stderr, "hipPointerSetAttribute(dst_d) failed: %d\n", cr);
		return 1;
	}

	pin_params_dst.va = (__u64)dst_d;
	pin_params_dst.size = SURFACE_SIZE * sizeof(*dst_d);
	ret = ioctl(fd, RDMA_IOC_PIN, &pin_params_dst);
	if (ret != 0) {
		fprintf(stderr, "ioctl(RDMA_IOC_PIN dst) failed: ret=%d errno=%d\n", ret, errno);
		return 1;
	}

#if (SURFACE_W < 16) || (SURFACE_H < 16)
#error Grid and block sizes must be shrunk for small surfaces
#endif
#if (SURFACE_W & 15) || (SURFACE_H & 15)
#error Grid and block sizes are not a multiple of the surface size
#endif
	dim3 dimGrid(SURFACE_W / 16, SURFACE_H / 16);
	dim3 dimBlock(16, 16);
	fill_surface<<<dimGrid, dimBlock>>>(src_d, 0);
	fill_surface<<<dimGrid, dimBlock>>>(dst_d, 0xffffffffU);

	ce = hipDeviceSynchronize();
	if (ce != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize() failed: %d\n", ce);
		return 1;
	}
	
	// Breakout here, as SGDMA not implemented
	glutMainLoop();
	return 1;

	dma_params.src = pin_params_src.handle;
	dma_params.dst = pin_params_dst.handle;
	dma_params.len = SURFACE_SIZE * sizeof(*src_d);
	dma_params.flags = RDMA_H2C2H_DMA_FLAG_SRC_IS_CUDA |
		RDMA_H2C2H_DMA_FLAG_DST_IS_CUDA;
	ret = ioctl(fd, RDMA_IOC_H2C2H_DMA, &dma_params);
	if (ret != 0) {
		fprintf(stderr, "ioctl(DMA) failed: %d\n", ret);
		return 1;
	}

	/*
	 * dGPU on x86 does not allow GPUDirect RDMA on host pinned memory
	 * (hipMalloc), so we must allocate device memory, and manually copy
	 * it to the host for validation.
	 */
#ifdef NV_BUILD_DGPU
	ce = hipHostMalloc(&dst_cpu, SURFACE_SIZE * sizeof(*dst_cpu), 0);
	if (ce != hipSuccess) {
		fprintf(stderr, "hipHostMalloc(dst_cpu) failed\n");
		return 1;
	}
	ce = hipMemcpy(dst_cpu, dst_d, SURFACE_SIZE * sizeof(*dst_cpu), hipMemcpyDeviceToHost);
	if (ce != hipSuccess) {
		fprintf(stderr, "hipMemcpy() failed: %d\n", ce);
		return 1;
	}
#else
	dst_cpu = dst_d;
#endif

	ret = 0;
	for (y = 0; y < SURFACE_H; y++) {
		for (x = 0; x < SURFACE_W; x++) {
			uint32_t expected = DATA(x, y);
			uint32_t offset = OFFSET(x, y);
			uint32_t actual = dst_cpu[offset];
			if (actual != expected) {
				fprintf(stderr,
					"dst[0x%x] is 0x%x not 0x%x\n",
					offset, actual, expected);
				ret = 1;
			}
		}
	}
	if (ret)
		return 1;

#ifdef NV_BUILD_DGPU
	ce = hipHostFree(dst_cpu);
	if (ce != hipSuccess) {
		fprintf(stderr, "hipHostFree(dst_cpu) failed: %d\n", ce);
		return 1;
	}
#endif

	unpin_params_dst.handle = pin_params_dst.handle;
	ret = ioctl(fd, RDMA_IOC_UNPIN, &unpin_params_dst);
	if (ret != 0) {
		fprintf(stderr, "ioctl(RDMA_IOC_UNPIN dst) failed: %d\n", ret);
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipFree(dst_d);
#else
	ce = hipHostFree(dst_d);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Free of dst_d failed: %d\n", ce);
		return 1;
	}

	unpin_params_src.handle = pin_params_src.handle;
	ret = ioctl(fd, RDMA_IOC_UNPIN, &unpin_params_src);
	if (ret != 0) {
		fprintf(stderr, "ioctl(RDMA_IOC_UNPIN src) failed: %d\n", ret);
		return 1;
	}

#ifdef NV_BUILD_DGPU
	ce = hipFree(src_d);
#else
	ce = hipHostFree(src_d);
#endif
	if (ce != hipSuccess) {
		fprintf(stderr, "Free of src_d failed: %d\n", ce);
		return 1;
	}

	ret = close(fd);
	if (ret < 0) {
		perror("close() failed");
		return 1;
	}

	return 0;
}

